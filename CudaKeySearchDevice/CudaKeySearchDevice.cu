#include "hip/hip_runtime.h"
#include "CudaKeySearchDevice.h"
#include "KeySearchTypes.h"
#include "ptx.cuh"
#include "secp256k1.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "ripemd160.cuh"
#include "sha256.cuh"

#include "secp256k1.h"

#include "CudaAtomicList.cuh"
#include "CudaDeviceKeys.cuh"
#include "CudaHashLookup.cuh"

__constant__ unsigned int _INC_X[8];

__constant__ unsigned int _INC_Y[8];

__constant__ unsigned int* _CHAIN[1];

static unsigned int* _chainBufferPtr = nullptr;


__device__ void doRMD160FinalRound(const unsigned int hIn[5], unsigned int hOut[5])
{
    const unsigned int iv[5] = {
        0x67452301,
        0xefcdab89,
        0x98badcfe,
        0x10325476,
        0xc3d2e1f0};

    for (int i = 0; i < 5; i++) {
        hOut[i] = endian(hIn[i] + iv[(i + 1) % 5]);
    }
}


/**
 * Allocates device memory for storing the multiplication chain used in
 the batch inversion operation
 */
hipError_t allocateChainBuf(unsigned int count)
{
    hipError_t err = hipMalloc(&_chainBufferPtr, count * sizeof(unsigned int) * 8);

    if (err) {
        return err;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(_CHAIN), &_chainBufferPtr, sizeof(unsigned int*));
    if (err) {
        hipFree(_chainBufferPtr);
    }

    return err;
}

void cleanupChainBuf()
{
    if (_chainBufferPtr != nullptr) {
        hipFree(_chainBufferPtr);
        _chainBufferPtr = nullptr;
    }
}

/**
 *Sets the EC point which all points will be incremented by
 */
hipError_t setIncrementorPoint(const secp256k1::uint256& x, const secp256k1::uint256& y)
{
    unsigned int xWords[8];
    unsigned int yWords[8];

    x.exportWords(xWords, 8, secp256k1::uint256::BigEndian);
    y.exportWords(yWords, 8, secp256k1::uint256::BigEndian);

    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_INC_X), xWords, sizeof(unsigned int) * 8);
    if (err) {
        return err;
    }

    return hipMemcpyToSymbol(HIP_SYMBOL(_INC_Y), yWords, sizeof(unsigned int) * 8);
}


__device__ void hashPublicKey(const unsigned int* x, const unsigned int* y, unsigned int* digestOut)
{
    unsigned int hash[8];

    sha256PublicKey(x, y, hash);

    // Swap to little-endian
    for (int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}

__device__ void hashPublicKeyCompressed(const unsigned int* x, unsigned int yParity, unsigned int* digestOut)
{
    unsigned int hash[8];

    sha256PublicKeyCompressed(x, yParity, hash);

    // Swap to little-endian
    for (int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}


__device__ void setResultFound(int idx, bool compressed, unsigned int x[8], unsigned int y[8], unsigned int digest[5])
{
    CudaDeviceResult r;

    r.block = blockIdx.x;
    r.thread = threadIdx.x;
    r.idx = idx;
    r.compressed = compressed;

    for (int i = 0; i < 8; i++) {
        r.x[i] = x[i];
        r.y[i] = y[i];
    }

    doRMD160FinalRound(digest, r.digest);

    atomicListAdd(&r, sizeof(r));
}

__device__ void doIteration(int pointsPerThread, int compression)
{
    unsigned int* chain = _CHAIN[0];
    unsigned int* xPtr = ec::getXPtr();
    unsigned int* yPtr = ec::getYPtr();

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0, 0, 0, 0, 0, 0, 0, 1};
    for (int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];

        unsigned int digest[5];

        readInt(xPtr, i, x);

        if (compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
            unsigned int y[8];
            readInt(yPtr, i, y);

            hashPublicKey(x, y, digest);

            if (checkHash(digest)) {
                setResultFound(i, false, x, y, digest);
            }
        }

        if (compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {
            hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

            if (checkHash(digest)) {
                unsigned int y[8];
                readInt(yPtr, i, y);
                setResultFound(i, true, x, y, digest);
            }
        }

        beginBatchAdd(_INC_X, x, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for (int i = pointsPerThread - 1; i >= 0; i--) {
        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAdd(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);
    }
}

__device__ void doIterationWithDouble(int pointsPerThread, int compression)
{
    unsigned int* chain = _CHAIN[0];
    unsigned int* xPtr = ec::getXPtr();
    unsigned int* yPtr = ec::getYPtr();

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0, 0, 0, 0, 0, 0, 0, 1};
    for (int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];

        unsigned int digest[5];

        readInt(xPtr, i, x);

        // uncompressed
        if (compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
            unsigned int y[8];
            readInt(yPtr, i, y);
            hashPublicKey(x, y, digest);

            if (checkHash(digest)) {
                setResultFound(i, false, x, y, digest);
            }
        }

        // compressed
        if (compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {
            hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

            if (checkHash(digest)) {
                unsigned int y[8];
                readInt(yPtr, i, y);

                setResultFound(i, true, x, y, digest);
            }
        }

        beginBatchAddWithDouble(_INC_X, _INC_Y, xPtr, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for (int i = pointsPerThread - 1; i >= 0; i--) {
        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAddWithDouble(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);
    }
}

/**
* Performs a single iteration
*/
__global__ void _stepKernel(int points, int compression)
{
    doIteration(points, compression);
}

__global__ void _stepKernelWithDouble(int points, int compression)
{
    doIterationWithDouble(points, compression);
}