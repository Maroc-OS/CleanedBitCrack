#include "hip/hip_runtime.h"
#include "cudabridge.h"


__global__ void _stepKernel(int points, int compression);
__global__ void _stepKernelWithDouble(int points, int compression);

void callKeyFinderKernel(int blocks, int threads, int points, bool useDouble, int compression)
{
    if (useDouble) {
        _stepKernelWithDouble<<<blocks, threads>>>(points, compression);
    } else {
        _stepKernel<<<blocks, threads>>>(points, compression);
    }
    waitForKernel();
}


void waitForKernel()
{
    // Check for kernel launch error
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw cuda::CudaException(err);
    }

    // Wait for kernel to complete
    err = hipDeviceSynchronize();
    fflush(stdout);
    if (err != hipSuccess) {
        throw cuda::CudaException(err);
    }
}